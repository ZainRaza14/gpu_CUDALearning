// Cuda Vec Add 


#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdio.h>
#include <math.h>

// Each thread produces one element of the output matrix
__global__ void vecAdd_1(double *a, double *b, double *c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < n)
	{
		c[id] = a[id] + b[id];
	}

}


// Each thread produces one row of the output matrix
__global__ void vecAdd_2(double *a, double *b, double *c, int n)
{
	int id = threadIdx.x;
	int id_1;

	for(int i = 0; i < n; i++)
	{
		id_1 = id * n + i; 

		c[id_1] = a[id_1] + b[id_1];

	}
}


// Each thread produces one column of the output matrix
__global__ void vecAdd_3(double *a, double *b, double *c, int n)
{
	int id = threadIdx.x;
	int id_1;

	for(int i = 0; i < n; i++)
	{
		id_1 = id + i * n; 
		
		c[id_1] = a[id_1] + b[id_1];

	}

}


int main(int argc, char* argv[])
{
	int n = 1024;
    

    // Host vectors
	double *h_a;
	double *h_b;
	double *h_c;

	//Device vectors

	double *d_a;
	double *d_b;
	double *d_c;

	// No of bytes
	size_t n_bytes = n*n*sizeof(double);

	// Allocating memory to host vectors
	h_a = (double*)malloc(n_bytes);
	h_b = (double*)malloc(n_bytes);
	h_c = (double*)malloc(n_bytes);

	// Allocating memory to device vectors
	hipMalloc(&d_a, n_bytes);
	hipMalloc(&d_b, n_bytes);
	hipMalloc(&d_c, n_bytes);

	// Initializing values of host vectors randomly


	for(int i = 0; i < n*n; i++)
	{
		h_a[i] = sin(i) * sin(i);
		h_b[i] = sin(i) * sin(i);
	}

	// Copying values from host to device

	hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;

	blockSize = 1024;

	gridSize = (int)ceil(float(n/blockSize));

	// Starting the first kernel
	//vecAdd_1<<< n , blockSize>>>(d_a, d_b, d_c, n);

	// Starting the second kernel
	//vecAdd_2<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

	// Starting the third kernel
	vecAdd_3<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

	// Copying back from Device to Host
	hipMemcpy(h_c, d_c, n_bytes, hipMemcpyDeviceToHost);

	double sum = 0;


	for(int j = 0; j < n; j++)
	{
		sum += h_c[j];
	}
	printf("Final result is: %f\n", sum/(double)n);

	// printing values of first input matrix
	for(int i=0; i< 25; i++)  
    {  
        printf("%lf  ", h_a[i] );  
    }

    printf("\n \n");

    //printing values of second input matrix
    for(int i=0; i< 25; i++)  
    {  
        printf("%lf  ", h_b[i]);  
    }  

    printf("\n \n");

    //printing values of output matrix
	for(int i=0; i< 25; i++)  
    {  
        printf("%lf  ", h_c[i]);  
    } 

    printf("\n \n");

	hipFree(d_a);
    hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

}