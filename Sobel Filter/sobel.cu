#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
#include"kernel.cu"

#define DEFAULT_THRESHOLD  8000

#define DEFAULT_FILENAME "BWstop-sign.ppm"

#define BLOCK_SIZE 32 

#define TILE_SIZE 32


// Using from Previous Home Work
template<class T, class S>
inline bool
compareData(const T *reference, const T *data, const unsigned int len,
            const S epsilon, const float threshold)
{
    //assert(epsilon >= 0);
    if( epsilon < 0)
      exit(3);

    bool result = true;
    unsigned int error_count = 0;

    for (unsigned int i = 0; i < len; ++i) 
    {
        float diff = (float)reference[i] - (float)data[i];
        bool comp = (diff <= epsilon) && (diff >= -epsilon);
        result &= comp;

        error_count += !comp;

#if _DEBUG

        if (! comp)
        {
            std::cerr << "ERROR, i = " << i << ",\t "
                      << reference[i] << " / "
                      << data[i]
                      << " (reference / data)\n";
        }

#endif
    }

    if (threshold == 0.0f)
    {
        return (result) ? true : false;
    }
    else
    {
        if (error_count)
        {
            printf("%4.2f(%%) of bytes mismatched (count=%d)\n", (float)error_count*100/(float)len, error_count);
        }

        return (len*threshold > error_count) ? true : false;
    }
}





unsigned int *read_ppm( char *filename, int &xsize, int &ysize, int &maxval ){
  
  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  FILE *fp;

  fprintf(stderr, "read_ppm( %s )\n", filename);
  fp = fopen( filename, "rb");
  if (!fp) 
  {
    fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
    return NULL; // fail 
  }

  char chars[1024];
  //int num = read(fd, chars, 1000);
  int num = fread(chars, sizeof(char), 1000, fp);

  if (chars[0] != 'P' || chars[1] != '6') 
  {
    fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
    return NULL;
  }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line! 
  {
    ptr = 1 + strstr(ptr, "\n");
  }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);  
  xsize = width;
  ysize = height;
  maxval = maxvalue;
  
  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if ((maxval) > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }

  // really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data. 
  sprintf(duh, "%d", xsize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d", ysize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d", maxval);
  line = strstr(line, duh);
  
  fprintf(stderr, "%s found at offset %ld\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  //lseek(fd, offset, SEEK_SET); // move to the correct offset
  fseek(fp, offset, SEEK_SET); // move to the correct offset
  //long numread = read(fd, buf, bufsize);
  long numread = fread(buf, sizeof(char), bufsize, fp);
  fprintf(stderr, "Texture %s   read %ld of %d bytes\n", filename, numread, bufsize); 

  fclose(fp);
  
  int pixels = (xsize) * (ysize);
  for (int i=0; i<pixels; i++) 
    pic[i] = (int) buf[3*i];  // red channel
  
  return pic; // success
}




void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic) 
{
  FILE *fp;
  //int x,y;
  
  fp = fopen(filename, "wb");
  if (!fp) 
  {
    //fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
    exit(-1); 
  }
  
  fprintf(fp, "P6\n"); 
  fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
  
  int numpix = xsize * ysize;
  for (int i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc); 
  }

  fclose(fp);
}

	
main( int argc, char **argv )
{

  int thresh = DEFAULT_THRESHOLD;
  
  char *filename;
 
  filename = strdup( DEFAULT_FILENAME);
  
  if (argc > 1) 
  {
    if (argc == 3)  
    { 
      // filename AND threshold
      filename = strdup( argv[1]);
      thresh = atoi( argv[2] );
    }
    
    if (argc == 2) 
    { 
      // default file but specified threshhold

      thresh = atoi( argv[1] );
    
    }

    fprintf(stderr, "file %s    threshold %d\n", filename, thresh); 
  }


  int xsize, ysize, maxval;
 
  unsigned int *pic = read_ppm( filename, xsize, ysize, maxval ); 
  
  unsigned int *pic_D;

  int numbytes =  xsize * ysize * 1 * sizeof( int );
  
  int *gpu_finalRes = (int *) malloc( numbytes );
  
  int *result = (int *) malloc( numbytes );

// CPU

  int i, j, magnitude, sum1, sum2;
  
  int *pointer_Res = result;

  for (int col=0; col<ysize; col++) 
  {
    for (int row=0; row<xsize; row++) 
    {
      *pointer_Res++ = 0;
    }
  
  }

  for (i = 1;  i < ysize - 1; i++) {
    for (j = 1; j < xsize -1; j++) {

      int offset = i*xsize + j;

      sum1 =  pic[ xsize * (i-1) + j+1 ] -     pic[ xsize*(i-1) + j-1 ]
        + 2 * pic[ xsize * (i)   + j+1 ] - 2 * pic[ xsize*(i)   + j-1 ]
        +     pic[ xsize * (i+1) + j+1 ] -     pic[ xsize*(i+1) + j-1 ];

      sum2 = pic[ xsize * (i-1) + j-1 ] + 2 * pic[ xsize * (i-1) + j ]  + pic[ xsize * (i-1) + j+1 ]
            - pic[xsize * (i+1) + j-1 ] - 2 * pic[ xsize * (i+1) + j ] - pic[ xsize * (i+1) + j+1 ];

      magnitude =  sum1*sum1 + sum2*sum2;

      if (magnitude > thresh)
        result[offset] = 255;
      else
        result[offset] = 0;
    }
  }

  write_ppm( "result_cpu.ppm", xsize, ysize, 255, result);


// GPU

  int *output_d;
 
  hipMalloc ((void **)&output_d, numbytes);
  
  hipMalloc ((void **)&pic_D, numbytes);

  hipMemcpy( pic_D, pic, numbytes, hipMemcpyHostToDevice);

  dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

  dim3 blockPerGrid((xsize/BLOCK_SIZE) + 1, (ysize/BLOCK_SIZE) + 1, 1); 

  sobel<<<blockPerGrid, threadPerBlock>>>(xsize, ysize, thresh, pic_D, output_d);
    
  hipDeviceSynchronize();
    
  hipMemcpy( gpu_finalRes, output_d, numbytes, hipMemcpyDeviceToHost);
    
  write_ppm( "result_gpu.ppm", xsize, ysize, 255, gpu_finalRes);

  bool res = compareData(result, gpu_finalRes, xsize * ysize * 1, 0.01f, 0.0f);

  printf("Test %s\n", (false == res) ? "FAILED" : "PASSED");
  
  hipFree(output_d);
    
  free(result);
    
  free(gpu_finalRes);

}

